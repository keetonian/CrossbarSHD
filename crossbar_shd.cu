#include "hip/hip_runtime.h"
/* 
 * CrossbarSHD
 * Software to simulate the CrossbarSHD algorithm
 *
 * output: Read_Name    0/16    Chromosome_name   Index_in_chromosome   Error_count
 * 
 * crossbar_shd.cpp
 * Author: Keeton Hodgson
 * Last modified: 1/27/2017
 */

#include <stdio.h>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <sstream>
#include "hip/hip_runtime.h"
#include ""

#include "compare.h"
#include "reference.h"
#include "commandline.h"

using namespace std;

void read_compare_func16(string header, string read_line, string chrom_name, vector<unsigned short> * ref, int shift, int threshold, int seed);
void read_compare_func4(int id, string header, string read_line, string chrom_name, vector<unsigned char> * ref, int shift, int threshold);

void CompareRead16(char* read_file, char* reference_file, int shift, int threshold);
void CompareRead4(char* read_file, char* reference_file, int shift, int threshold);

/**
 * Function to do the 16-bit comparison.
 * ref: pointer to reference genome (large compared to read). Format: 16-bit 1-hot encodings.
 * read: pointer to read sequence. Again, 16-bit 1-hot encodings.
 */
__global__ void compare16(const unsigned short * ref, const unsigned short * read, char * output, int numElements, int rIndex) {

  __shared__ short rd[1024];

  int i = threadIdx.x + threadIdx.y + blockIdx.x*blockDim.x;
  int ii = threadIdx.x + threadIdx.y*blockDim.x;

  rd[ii] = 0;
  if(i < numElements){
    if(read[threadIdx.x+rIndex]&ref[i])
      rd[ii] = 1;
  }

  __syncthreads();

  if(threadIdx.x % 2 == 0 && threadIdx.x + 1 < blockDim.x){
    rd[ii] += rd[ii+1];
  }
  __syncthreads();
  if(threadIdx.x % 4 == 0 && threadIdx.x + 2 < blockDim.x){
    rd[ii] += rd[ii+2];
  }
  __syncthreads();
  if(threadIdx.x % 8 == 0 && threadIdx.x + 4 < blockDim.x){
    rd[ii] += rd[ii+4];
  }
  __syncthreads();
  if(threadIdx.x % 16 == 0 && threadIdx.x + 8 < blockDim.x){
    rd[ii] += rd[ii+8];
  }
  __syncthreads();
  if(threadIdx.x % 32 == 0 && threadIdx.x + 16 < blockDim.x){
    rd[ii] += rd[ii+16];
  }
  __syncthreads();

  if(threadIdx.x == 0)
    output[i] = rd[ii];
}

char* sequence = 0;
int seed_size = 32;

int main(int argc, char** argv)
{
  if(argc < 2) {
    cerr << "Error: no arguments specified.\n";
    return 1;
  }

  char* reference_file = 0;
  char* read_file = 0;

  // Set some default parameters
  int shift = 0;
  int encoding = 1;
  int threshold = 0;

  if(CommandLineOptions(argc, argv, &shift, &encoding, &threshold, &reference_file, &read_file, &sequence, &seed_size))
    exit(0);

  //cout<<"shift: "<< shift<<endl;
  //cout<<"threshold: "<< threshold<<endl;

  if(!reference_file || !read_file) {
    cerr<< "Need to specify a reference and read file.\n";
    return 1;
  } 

  if(encoding){
    //cout<<"16 Bit encodings.\n";
    CompareRead16(read_file, reference_file, shift, threshold);
  }
  else{
    //cout<<"4 Bit encodings.\n";
    CompareRead4(read_file, reference_file, shift, threshold);
  }

  return 0;
}

/*
 * Compares each read in the read_file to the reference genome according to the shift and threshold
 * 16 bit version
 */
void CompareRead16(char* read_file, char* reference_file, int shift, int threshold) {

  // Convert the reference to 16-bit vectors
  vector<vector<unsigned short> * > ref(0);
  vector<string> ref_names(0);
  ref.reserve(64);
  ref_names.reserve(64);
  PrepareReference16(reference_file, &ref, &ref_names);


  // Get the starting read if it exists.
  string str;
  if(sequence != 0)
    str = string(sequence);

  // Do the comparisons one chromosome at a time to be more memory efficient
  for(unsigned int chromosome = 0; chromosome < ref.size(); chromosome++){

    string chrom_name = ref_names.at(chromosome);

    // Prepare sizes of elements
    int numElements=ref.at(chromosome)->size();
    size_t size = numElements*sizeof(short);
    size_t size_out = numElements*sizeof(char);
    char* hOutput = (char*)malloc(size_out);


    // Reserve space for the Reference, Output arrays in the GPU
    hipError_t err = hipSuccess;
    unsigned short *cRef = NULL;
    err = hipMalloc((void **) &cRef, size);
    if(err != hipSuccess){
      cerr << "Failed to allocate device vector ref (error code " << hipGetErrorString(err) << ")!" << endl;
      exit(1);
    }

    char* cOutput = NULL;
    err = hipMalloc((void **) &cOutput, size_out);
    if(err != hipSuccess) {
      cerr << "CudaFailure on cOutput" << hipGetErrorString(err) << endl;
      exit(1);
    }

    // Copy the reference chromosome into the GPU
    err = hipMemcpy(cRef, (unsigned short*)(&(ref.at(chromosome)->at(0))), size, hipMemcpyHostToDevice);
    if(err != hipSuccess) {
      cerr << "Cuda Failure on copying cRef" << endl;
      exit(1);
    }

    // Prepare the thread dimensions
    dim3 threadsPerBlock(seed_size, seed_size);
    int blocksPerGrid= (numElements + seed_size)/(seed_size);

    // Extract the reads (name and sequence) from the read file
    string read_line;
    ifstream file(read_file);
    if(file.is_open()) {

      // Step through the read sequences
      while(getline(file, read_line)) {
	
	// Extract the name
	if(read_line.size() > 2 && read_line[0] == '@' && read_line[1] == 'E' && read_line[2] == 'R') {
	  char c = read_line[1];
	  string name = "";
	  int cindex = 1;
	  while(c != ' ') {
	    name+=c;
	    cindex++;
	    c = read_line[cindex];
	  }
	  // s+='\t';
	  if(sequence != 0){
	    if(str == name)
	      sequence = 0;
	    else
	      continue;
	  }

	  // Get the sequence
	  getline(file, read_line);

	  // Prepare the sequences to be used by the GPU
	  int read_size = read_line.size()-1;
	  vector<unsigned short> readv_temp, read_inverse_temp;

	  // Allocate space for the arrays
	  unsigned short * readv, * read_inverse;
	  readv = (unsigned short *)malloc((read_size)*sizeof(short));
	  read_inverse = (unsigned short *)malloc((read_size)*sizeof(short));
	  size_t size_read = (read_size)*sizeof(short);
	  for(int j = 0; j < read_size; j++) {
	    readv_temp.push_back(ConvertCharacters16(read_line[j], read_line[j+1]));
	    read_inverse_temp.push_back(ConvertInverseCharacters16(read_line[read_size-(j)], read_line[read_size-(j+1)]));
	  }

	  // Copy the data into the arrays, using appropriate shift distance
	  for(int i = 0; i < readv_temp.size(); i++){
	    unsigned short a = readv_temp.at(i);
	    unsigned short b = read_inverse_temp.at(i);
	    for(int j = 0; j <= shift; j++){
	      if(i>=j){
		a = a | readv_temp.at(i-j);
		b = b | read_inverse_temp.at(i-j);
	      }
	      if((i+j)<readv_temp.size()){
		a = a | readv_temp.at(i+j);
		b = b | read_inverse_temp.at(i+j);
	      }
	    }
	    readv[i] = a;
	    read_inverse[i] = b;
	  }

	  // Allocate space on GPU for the forward/reverse inverse reads
	  unsigned short *cRead = NULL;
	  err = hipMalloc((void **) &cRead, size_read);
	  if(err != hipSuccess) {
	    cerr << "Cuda failure on cRead" << endl;
	    exit(1);
	  }

	  unsigned short *cInverse = NULL;
	  err = hipMalloc((void **) &cInverse, size_read);
	  if(err != hipSuccess) {
	    cerr << "Cuda failure on cInverse" << endl;
	    exit(1);
	  }

	  err = hipMemcpy(cRead, readv, size_read, hipMemcpyHostToDevice);
	  if(err != hipSuccess) {
	    cerr << "Cuda failure on copying cRead " << hipGetErrorString(err) << endl;
	    exit(1);
	  }

	  err = hipMemcpy(cInverse, read_inverse, size_read, hipMemcpyHostToDevice);
	  if(err != hipSuccess) {
	    cerr << "Cuda failure on copying cInverse " << hipGetErrorString(err) << endl;
	    exit(1);
	  }

	  // Do the calculations for each seed
	  for(int i = 0; i < (read_size)/seed_size; i++){

	    compare16<<<blocksPerGrid,threadsPerBlock>>>(cRef, cRead, cOutput, numElements, i*seed_size);
	    err = hipGetLastError();
	    if(err != hipSuccess){
	      cout << "Failure on cuda computation: " << hipGetErrorString(err) << endl;
	      exit(1);
	    }

	    err = hipMemcpy(hOutput, cOutput, size_out, hipMemcpyDeviceToHost);
	    if(err != hipSuccess){
	      cerr << "Failure on copying output" << hipGetErrorString(err) << endl;
	      exit(1);
	    }

	    // Print the results
	    for(int ii = 0; ii < numElements; ii++)
	      if(hOutput[ii] >= seed_size-threshold)
		cout << name << '\t' << seed_size << '\t' << i << '\t' << "0" << '\t' << chrom_name << '\t' << ii << '\t' << (int)hOutput[ii] << endl;

	  }

	  // Do the calculations for each rinverse seed
	  for(int i = 0; i < (read_size)/seed_size; i++){

	    compare16<<<blocksPerGrid,threadsPerBlock>>>(cRef, cInverse, cOutput, numElements, i*seed_size);
	    err = hipGetLastError();
	    if(err != hipSuccess){
	      cout << "Failure on cuda computation: " << hipGetErrorString(err) << endl;
	      exit(1);
	    }

	    err = hipMemcpy(hOutput, cOutput, size_out, hipMemcpyDeviceToHost);
	    if(err != hipSuccess){
	      cerr << "Failure on copying output" << hipGetErrorString(err) << endl;
	      exit(1);
	    }

	    // Print the results
	    for(int ii = 0; ii < numElements; ii++)
	      if(hOutput[ii] >= seed_size-threshold)
		cout << name << '\t' << seed_size << '\t' << i << '\t' << "16" << '\t' << chrom_name << '\t' << ii << '\t' << (int)hOutput[ii] << endl;
	  }

	  err = hipFree(cRead);
	  if(err != hipSuccess) {
	    cerr << "Failure freeing read" << endl;
	    exit(1);
	  }
	  err = hipFree(cInverse);
	  if(err != hipSuccess) {
	    cerr << "failure freeing cinverse" << endl;
	    exit(1);
	  }
	  free(readv);
	  free(read_inverse);

	}
      }
    }
    else {
      cerr << "Unable to open reference file \n";
      exit(1);
    }

    cout << "HERE" << endl;

    // Free all GPU memory
    err = hipFree(cRef);
    if(err != hipSuccess) {
      cerr << "Failure freeing reference" << endl;
      exit(1);
    }

    err = hipFree(cOutput);
    if(err != hipSuccess) {
      cerr << "Failure freeing output" << endl;
      exit(1);
    }

    free(hOutput);
  }
} 

/*
 * Compares each read in the read_file to the reference genome according o the shift and threshold
 * 4 bit version
 */
void CompareRead4(char* read_file, char* reference_file, int shift, int threshold) {
  vector<vector<unsigned char>*> ref(0);
  vector<string> ref_names(0);
  ref.reserve(64);
  ref_names.reserve(64);
  PrepareReference4(reference_file, &ref, &ref_names);
  //ctpl::thread_pool p(8);

  // Set up and read from the file with the read sequences
  string read_line;
  ifstream file(read_file);
  if(file.is_open()) {

    // Step through the read sequences
    while(getline(file, read_line)) {
      if(read_line.size() > 2 && read_line[0] == '@' && read_line[1] == 'E' && read_line[2] == 'R') {
	char c = read_line[1];
	string s = "";
	int cindex = 1;
	while(c != ' ') {
	  s+=c;
	  cindex++;
	  c = read_line[cindex];
	}
	//s+='\t';

	getline(file, read_line);
	for(unsigned int i = 0; i < ref.size(); i++) {
	  //      p.push(read_compare_func4, s, read_line, ref_names.at(i), ref.at(i), shift, threshold);
	}
      }
    }
  }
  else {
    cerr << "Unable to open reference file \n";
    exit(1);
  }
} 

/*
 * Helper function for CompareRead4. THis function is passed into threads to be run
 */
void read_compare_func4(int id, string header, string read_line, string chrom_name, vector<unsigned char> * ref, int shift, int threshold) {
  int j = 0;
  // Save read sequence
  //header += read_line + '\n';

  std::vector<unsigned char> readv(0);
  std::vector<unsigned char> read_inverse(0);
  int read_size = read_line.size();
  for(; j < read_size-1; j++) {
    readv.push_back(ConvertCharacter4(read_line[j]));
    read_inverse.push_back(ConvertCharacterInverse4(read_line[read_size-(j+1)]));
  }

  int num_matches = 0;
  string res = compare4(ref, readv, read_inverse, shift, threshold, &num_matches, header, chrom_name);

  if(res.size() > 1) {
    //if(report_total_matches)
    //cout<<res<<"TOTAL MATCHES: " << num_matches << '\n' << endl;
    //else
    cout<<res.substr(0, res.size() - 1)<<endl;
  }
}
