#include "hip/hip_runtime.h"
/* 
 * CrossbarSHD
 * Software to simulate the CrossbarSHD algorithm
 *
 * output: Read_Name    0/16    Chromosome_name   Index_in_chromosome   Error_count
 * 
 * crossbar_shd.cpp
 * Author: Keeton Hodgson
 * Last modified: 1/27/2017
 */

#include <stdio.h>
#include <vector>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <sstream>
#include "hip/hip_runtime.h"

#include "compare.h"
#include "reference.h"
#include "commandline.h"

using namespace std;

void read_compare_func16(string header, string read_line, string chrom_name, vector<unsigned short> * ref, int shift, int threshold);
void read_compare_func4(int id, string header, string read_line, string chrom_name, vector<unsigned char> * ref, int shift, int threshold);

void CompareRead16(char* read_file, char* reference_file, int shift, int threshold);
void CompareRead4(char* read_file, char* reference_file, int shift, int threshold);

/**
 * Function to do the 16-bit comparison.
 * ref: pointer to reference genome (large compared to read). Format: 16-bit 1-hot encodings.
 * read: pointer to read sequence. Again, 16-bit 1-hot encodings.
 */
__global__ void compare16(const unsigned short * ref, const unsigned short * read, char * output, int numElements) {

  __shared__ int s[64];

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i < numElements && read[threadIdx.x]&ref[i])
    s[threadIdx.x] = 1;
  else
    s[threadIdx.x] = 0;

  __syncthreads();

  if(threadIdx.x % 2 == 0 && threadIdx.x + 1 < blockDim.x)
    s[threadIdx.x] += s[threadIdx.x+1];
  __syncthreads();
  if(threadIdx.x % 4 == 0 && threadIdx.x + 2 < blockDim.x)
    s[threadIdx.x] += s[threadIdx.x+2];
  __syncthreads();
  if(threadIdx.x % 8 == 0 && threadIdx.x + 4 < blockDim.x)
    s[threadIdx.x] += s[threadIdx.x+4];
  __syncthreads();
  if(threadIdx.x % 16 == 0 && threadIdx.x + 8 < blockDim.x)
    s[threadIdx.x] += s[threadIdx.x+8];
  __syncthreads();
  if(threadIdx.x % 32 == 0 && threadIdx.x + 16 < blockDim.x)
    s[threadIdx.x] += s[threadIdx.x+16];
  __syncthreads();

  if(threadIdx.x == 0)
    //output[blockIdx.x] = s[threadIdx.x];
    output[blockIdx.x] = blockIdx.x;
}

char* sequence = 0;
int threads = 2;

int main(int argc, char** argv)
{
  if(argc < 2) {
    cerr << "Error: no arguments specified.\n";
    return 1;
  }

  char* reference_file = 0;
  char* read_file = 0;

  // Set some default parameters
  int shift = 0;
  int encoding = 1;
  int threshold = 0;
  
  if(CommandLineOptions(argc, argv, &shift, &encoding, &threshold, &reference_file, &read_file, &sequence, &threads))
    exit(0);

  //cout<<"shift: "<< shift<<endl;
  //cout<<"threshold: "<< threshold<<endl;

  if(!reference_file || !read_file) {
    cerr<< "Need to specify a reference and read file.\n";
    return 1;
  } 

  if(encoding){
    //cout<<"16 Bit encodings.\n";
    CompareRead16(read_file, reference_file, shift, threshold);
  }
  else{
    //cout<<"4 Bit encodings.\n";
    CompareRead4(read_file, reference_file, shift, threshold);
  }

  return 0;
}

/*
 * Compares each read in the read_file to the reference genome according to the shift and threshold
 * 16 bit version
 */
void CompareRead16(char* read_file, char* reference_file, int shift, int threshold) {
  vector<vector<unsigned short> * > ref(0);
  vector<string> ref_names(0);
  ref.reserve(64);
  ref_names.reserve(64);
  PrepareReference16(reference_file, &ref, &ref_names);
  //ctpl::thread_pool p(threads);
  string str;
  if(sequence != 0)
    str = string(sequence);

  // Set up and read from the file with the read sequences
  string read_line;
  ifstream file(read_file);
  if(file.is_open()) {

    // Step through the read sequences
    while(getline(file, read_line)) {
      if(read_line.size() > 2 && read_line[0] == '@' && read_line[1] == 'E' && read_line[2] == 'R') {
        char c = read_line[1];
        string name = "";
        int cindex = 1;
        while(c != ' ') {
          name+=c;
          cindex++;
          c = read_line[cindex];
        }
        // s+='\t';
        if(sequence != 0){
          if(str == name)
            sequence = 0;
          else
            continue;
        }


        getline(file, read_line);
        //for(unsigned int i = 0; i < ref.size(); i++){
          //p.push(read_compare_func16, name, read_line, ref_names.at(i), ref.at(i), shift, threshold);
	  read_compare_func16(name, read_line, ref_names.at(0), ref.at(0), shift, threshold);
        //}
      }
    }
  }
  else {
    cerr << "Unable to open reference file \n";
    exit(1);
  }
} 

/*
 * Helper function for CompareRead16. This function is passed into thread arguments
 */
void read_compare_func16(string header, string read_line, string chrom_name, vector<unsigned short> * ref, int shift, int threshold) {
  int j = 0;
  // Save read sequence
  //header += read_line + '\n';

  cout << "Converting string" << endl;
  std::vector<unsigned short> readv_temp, read_inverse_temp;
  unsigned short * readv, * read_inverse;
  readv = (unsigned short *)malloc((read_line.size()-1)*sizeof(short));
  read_inverse = (unsigned short *)malloc((read_line.size()-1)*sizeof(short));
  int read_size = read_line.size();
  for(; j < read_size-1; j++) {
    readv_temp.push_back(ConvertCharacters16(read_line[j], read_line[j+1]));
    read_inverse_temp.push_back(ConvertInverseCharacters16(read_line[read_size-(j+1)], read_line[read_size-(j+2)]));
  }

  cout << "Doing shift" << endl;
  for(int i = 0; i < readv_temp.size(); i++){
    unsigned short a = readv_temp.at(i);
    unsigned short b = read_inverse_temp.at(i);
    for(j = 0; j <= shift; j++){
      if(i>=j){
	a = a | readv_temp.at(i-j);
	b = b | read_inverse_temp.at(i-j);
      }
      if((i+j)<readv_temp.size()){
	a = a | readv_temp.at(i+j);
	b = b | read_inverse_temp.at(i+j);
      }
    }
    readv[i] = a;
    read_inverse[i] = b;
  }
  
  cout << "Starting cuda memory allocation" << endl;

  int numElements=ref->size();
  size_t size = numElements*sizeof(short);
  size_t size_out = numElements*sizeof(char);
  size_t size_read = (read_size-1)*sizeof(short);
  char * hOutput = (char*)malloc(size_out);

  hipError_t err = hipSuccess;
  unsigned short *cRef = NULL;
  err = hipMalloc((void **) &cRef, size);
  if(err != hipSuccess){
    cerr << "Failed to allocate device vector ref (error code " << hipGetErrorString(err) << ")!" << endl;
    exit(1);
  }

  unsigned short *cRead = NULL;
  err = hipMalloc((void **) &cRead, size_read);
  if(err != hipSuccess) {
    cerr << "Cuda failure on cRead" << endl;
    exit(1);
  }

  char * cOutput = NULL;
  err = hipMalloc((void **) &cOutput, size_out);
  if(err != hipSuccess) {
    cerr << "CudaFailure on cOutput" << endl;
    exit(1);
  }

  err = hipMemcpy(cRead, readv, size_read, hipMemcpyHostToDevice);
  if(err != hipSuccess) {
    cerr << "Cuda failure on copying cRead " << hipGetErrorString(err) << endl;
    exit(1);
  }

  cout << "Doing reference " << endl;

  err = hipMemcpy(cRef, (unsigned char *)(&(ref->at(0))), size, hipMemcpyHostToDevice);
  if(err != hipSuccess) {
    cerr << "Cuda Failure on copying cRef" << endl;
    exit(1);
  }

  int threadsPerBlock = 32;
  int blocksPerGrid = (numElements-threadsPerBlock) / threadsPerBlock;

  cout << "Starting cuda computation" << endl;

  cout << numElements << endl;
  cout << blocksPerGrid << endl;
  blocksPerGrid = blocksPerGrid>>6;
  compare16<<<blocksPerGrid, threadsPerBlock>>>(cRef, cRead, cOutput, numElements);
  err = hipGetLastError();
  if(err != hipSuccess){
    cout << "Failure on cuda computation: " << hipGetErrorString(err) << endl;
    exit(1);
  }

  cout << "Starting cuda freeing memory" << endl;

  err = hipMemcpy(hOutput, cOutput, size_out, hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    cerr << "Failure on copying output" << endl;
    exit(1);
  }

  err = hipFree(cRef);
  if(err != hipSuccess) {
    cerr << "Failure freeing reference" << endl;
    exit(1);
  }
  err = hipFree(cRead);
  if(err != hipSuccess) {
    cerr << "Failure freeing read" << endl;
    exit(1);
  }
  err = hipFree(cOutput);
  if(err != hipSuccess) {
    cerr << "Failure freeing output" << endl;
    exit(1);
  }

  for(int ii = 0; ii < numElements; ii++)
    if(hOutput[ii] >= 31)
      cout << (int)hOutput[ii] << endl;

  free(hOutput);
  free(readv);
  free(read_inverse);

}

/*
 * Compares each read in the read_file to the reference genome according o the shift and threshold
 * 4 bit version
 */
void CompareRead4(char* read_file, char* reference_file, int shift, int threshold) {
  vector<vector<unsigned char>*> ref(0);
  vector<string> ref_names(0);
  ref.reserve(64);
  ref_names.reserve(64);
  PrepareReference4(reference_file, &ref, &ref_names);
  //ctpl::thread_pool p(8);

  // Set up and read from the file with the read sequences
  string read_line;
  ifstream file(read_file);
  if(file.is_open()) {

    // Step through the read sequences
    while(getline(file, read_line)) {
      if(read_line.size() > 2 && read_line[0] == '@' && read_line[1] == 'E' && read_line[2] == 'R') {
        char c = read_line[1];
        string s = "";
        int cindex = 1;
        while(c != ' ') {
          s+=c;
          cindex++;
          c = read_line[cindex];
        }
        //s+='\t';

        getline(file, read_line);
        for(unsigned int i = 0; i < ref.size(); i++) {
    //      p.push(read_compare_func4, s, read_line, ref_names.at(i), ref.at(i), shift, threshold);
        }
      }
    }
  }
  else {
    cerr << "Unable to open reference file \n";
    exit(1);
  }
} 

/*
 * Helper function for CompareRead4. THis function is passed into threads to be run
 */
void read_compare_func4(int id, string header, string read_line, string chrom_name, vector<unsigned char> * ref, int shift, int threshold) {
  int j = 0;
  // Save read sequence
  //header += read_line + '\n';

  std::vector<unsigned char> readv(0);
  std::vector<unsigned char> read_inverse(0);
  int read_size = read_line.size();
  for(; j < read_size-1; j++) {
    readv.push_back(ConvertCharacter4(read_line[j]));
    read_inverse.push_back(ConvertCharacterInverse4(read_line[read_size-(j+1)]));
  }

  int num_matches = 0;
  string res = compare4(ref, readv, read_inverse, shift, threshold, &num_matches, header, chrom_name);

  if(res.size() > 1) {
  //if(report_total_matches)
    //cout<<res<<"TOTAL MATCHES: " << num_matches << '\n' << endl;
  //else
    cout<<res.substr(0, res.size() - 1)<<endl;
  }
}
